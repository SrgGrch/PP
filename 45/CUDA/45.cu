#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include <hip/hip_runtime.h>

#include <memory>
#include <conio.h>
#include <iostream>
#include <cassert>
#include <stdio.h>
#include <algorithm>


__global__ void findPrime(int n, int* x, int* y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

		for (int j = 3; j <= x[i] / 2; j++) {
			if (x[i] % j == 0) return;
		}
	

	int number = x[i];
	int digitsCounter = 0;

	while (number != 0) {
		digitsCounter++;
		number /= 10;
	}

	if (digitsCounter <= 2) return;

	//check end
	if ((x[i] % 10) <= 2) return;

	for (int j = 2; j <= (x[i] % 10)/2; j++) {
		if ((x[i] % 10) % j == 0) return;
	}

	int powRes = 10;

	for (int j = 0; j < digitsCounter-2; j++) {
		powRes *= 10;
	}

	// check start
	if ((x[i] / powRes) <= 2) return;

	for (int j = 2; j <= (x[i] / powRes) / 2; j++) {
		if ((x[i] / (powRes)) % j == 0) return;
	}

	powRes = 10;
	
	for (int j = 0; j < digitsCounter - 3; j++) {
		powRes *= 10;
	}
	// Check middle
	int middle = (x[i] / 10) % powRes;

	if (middle <= 2) return;

	for (int j = 2; j <= middle/2; j++) {
		if (middle % j == 0) return;
	}

	y[i] = 1;
}

int main(void)
{
	int N = 30109;
	int batchSize = 1024;
	int* x, * d_x, * y, * d_y;
	x = (int*)malloc(batchSize * sizeof(int));
	y = (int*)malloc(batchSize * sizeof(int));

	hipMalloc(&d_x, batchSize * sizeof(int));
	hipMalloc(&d_y, batchSize * sizeof(int));

	bool isPrimeFound = false;

	while (!isPrimeFound) {
		if (N % 2 == 0) N++;
		int num = 0;
		for (int i = 0; i < batchSize; i++) {
			x[i] = N + num;
			y[i] = 0;
			num += 2;
		}
	
		std::cout << "Searching in range: " << x[0] << " - " << x[batchSize - 1] << std::endl;

		hipMemcpy(d_x, x, batchSize * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_y, y, batchSize * sizeof(int), hipMemcpyHostToDevice);

		dim3 threadsPerBlock(32, 32);
		dim3 numBlocks(batchSize / threadsPerBlock.x, batchSize / threadsPerBlock.y);

		findPrime <<<numBlocks, threadsPerBlock>>> (N, d_x, d_y);
		
		hipMemcpy(y, d_y, batchSize * sizeof(int), hipMemcpyDeviceToHost);

		for (int i = 0; i < batchSize; i++) {
			if (y[i] == 1) {
				isPrimeFound = true;
				printf("%d ", x[i]);
				break;
			}
		}
		N = x[batchSize - 1];
	}

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
	//_getch();
}