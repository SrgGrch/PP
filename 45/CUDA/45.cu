#include <hip/hip_runtime.h>




#include <memory>
#include <iostream>
#include <cassert>
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <time.h>


/*
45.     ����� ���������� ������� �����, ������� ��������� N �����, ��� ������, �������� � 
��������� ��� ����������� ������������� �������� �������� ������� (��������, ����� 8353 � 83 5 3)
*/

__global__ void findPrime(int n, int* x, int* y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

		for (int j = 3; j <= x[i] / 2; j++) {
			if (x[i] % j == 0) return;
		}
	

	int number = x[i];
	int digitsCounter = 0;

	while (number != 0) {
		digitsCounter++;
		number /= 10;
	}

	if (digitsCounter <= 2) return;

	//check end
	if ((x[i] % 10) <= 2) return;

	for (int j = 2; j <= (x[i] % 10)/2; j++) {
		if ((x[i] % 10) % j == 0) return;
	}

	int powRes = 10;

	for (int j = 0; j < digitsCounter-2; j++) {
		powRes *= 10;
	}

	// check start
	if ((x[i] / powRes) <= 2) return;

	for (int j = 2; j <= (x[i] / powRes) / 2; j++) {
		if ((x[i] / (powRes)) % j == 0) return;
	}

	powRes = 10;
	
	for (int j = 0; j < digitsCounter - 3; j++) {
		powRes *= 10;
	}
	// Check middle
	int middle = (x[i] / 10) % powRes;

	if (middle <= 2) return;

	for (int j = 2; j <= middle/2; j++) {
		if (middle % j == 0) return;
	}

	y[i] = 1;
}

bool isPrime(int x)
{
	for (int j = 3; j <= x / 2; j++) {
		if (x % j == 0) return false;
	}


	int number = x;
	int digitsCounter = 0;

	while (number != 0) {
		digitsCounter++;
		number /= 10;
	}

	if (digitsCounter <= 2) return false;

	//check end
	if ((x % 10) <= 2) return false;

	for (int j = 2; j <= (x % 10) / 2; j++) {
		if ((x % 10) % j == 0) return false;
	}

	int powRes = 10;

	for (int j = 0; j < digitsCounter - 2; j++) {
		powRes *= 10;
	}

	// check start
	if ((x / powRes) <= 2) return false;

	for (int j = 2; j <= (x / powRes) / 2; j++) {
		if ((x / (powRes)) % j == 0) return false;
	}

	powRes = 10;

	for (int j = 0; j < digitsCounter - 3; j++) {
		powRes *= 10;
	}
	// Check middle
	int middle = (x / 10) % powRes;

	if (middle <= 2) return false;

	for (int j = 2; j <= middle / 2; j++) {
		if (middle % j == 0) return false;
	}

	return true;
}
	
int main(void)
{
	int Nstart;
		
	printf("������� ��������� �����: ");
	scanf("%d", &Nstart);
	int N = Nstart;
	int batchSize = 1024;
	int* x, * d_x, * y, * d_y;
	x = (int*)malloc(batchSize * sizeof(int));
	y = (int*)malloc(batchSize * sizeof(int));

	hipMalloc(&d_x, batchSize * sizeof(int));
	hipMalloc(&d_y, batchSize * sizeof(int));

	bool isPrimeFound = false;

	long double start = clock();

	while (!isPrimeFound) {
		if (N % 2 == 0) N++;
		int num = 0;
		for (int i = 0; i < batchSize; i++) {
			x[i] = N + num;
			y[i] = 0;
			num += 2;
		}
	
		std::cout << "Searching in range: " << x[0] << " - " << x[batchSize - 1] << std::endl;

		hipMemcpy(d_x, x, batchSize * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_y, y, batchSize * sizeof(int), hipMemcpyHostToDevice);

		dim3 threadsPerBlock(1024, 1, 1);
		dim3 numBlocks(batchSize / threadsPerBlock.x, 1, 1);

		findPrime <<<numBlocks, threadsPerBlock>>> (N, d_x, d_y);
		
		hipMemcpy(y, d_y, batchSize * sizeof(int), hipMemcpyDeviceToHost);

		for (int i = 0; i < batchSize; i++) {
			if (y[i] == 1) {
				isPrimeFound = true;
				printf(" ��������� �����: %d \n", x[i]);
				printf("GPU: %lf ms\n", (double)(clock() - start) / CLOCKS_PER_SEC * 1000);
				break;
			}
		}
		N = x[batchSize - 1];
	}

	start = clock();

	while (true) {
		if (Nstart % 2 == 0) N++;

		if (isPrime(Nstart)) {
			printf(" ��������� �����: %d \n", Nstart);
			printf("CPU: %lf ms\n", (double)(clock() - start) / CLOCKS_PER_SEC * 1000);
			break;
		}
		Nstart++;
	}


	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
}